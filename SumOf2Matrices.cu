#include "hip/hip_runtime.h"


#include <iostream>

__global__ void vectorAdd(int* A, int* B, int* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main() {
    // your code goes here

    int a[] = { 1,2,3 };
    int b[] = { 4,5,6 };
    int c[sizeof(a) / sizeof(int)] = { 0 };

    // create parameters into gpu
    int* cudaA = 0;
    int* cudaB = 0;
    int* cudaC = 0;

    // allocate memory into gpu
    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    // copy the vectors into gpu
    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

    vectorAdd<<<1, sizeof(a) / sizeof(int)>>> (cudaA, cudaB, cudaC);

    hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

    for (int i : c)
        std::cout << i << std::endl;

    // Free allocated memory
    hipFree(cudaA);
    hipFree(cudaB);
    hipFree(cudaC);

    return 0;
}
